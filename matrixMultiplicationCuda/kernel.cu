﻿
#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void matrixMultiplicationKernel(const double* a, const double* b, double* c, const size_t size)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    double sum = 0.0;
    if (col < size && row < size)
    {
        for (int i = 0; i < size; i++)
        {
            sum += a[row * size + i] * b[i * size + col];
        }
        c[row * size + col] = sum;
    }
}

int main()
{
    const size_t size = 1000;


    double* a = new double[size * size];
    double* b = new double[size * size];
    double* c = new double[size * size];

    for (int i = 0; i < size * size; ++i)
    {
        a[i] = 5.0;
        b[i] = 5.0;
        c[i] = 0.0;
    }

    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return 1;
    }

    const size_t sizeBytes = size * size * sizeof(double);
    double* aDevice;
    double* bDevice;
    double* cDevice;

    cudaStatus = hipMalloc((void**)&aDevice, sizeBytes);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return 1;
    }

   

    cudaStatus = hipMalloc((void**)&bDevice, sizeBytes);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return 1;
    }

    cudaStatus = hipMalloc((void**)&cDevice, sizeBytes);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return 1;
    }

    cudaStatus = hipMemcpy(aDevice, a, sizeBytes, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return 1;
    }

    hipMemcpy(bDevice, b, sizeBytes, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return 1;
    }


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    const int block_size = 32;
    dim3 dimBlock(block_size, block_size);
    dim3 dimGrid(size / block_size + 1, size / block_size + 1);

    hipEventRecord(start);
    matrixMultiplicationKernel<<<dimGrid, dimBlock>>>(aDevice, bDevice, cDevice, size);
    hipEventRecord(stop);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return -1;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        return 1;
    }

    cudaStatus = hipMemcpy(c, cDevice, sizeBytes, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return 1;
    }

    hipFree(aDevice);
    hipFree(bDevice);
    hipFree(cDevice);

    hipEventSynchronize(stop);
    hipDeviceSynchronize();

    float milliseconds;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("CUDA time simple (ms): %f\n", milliseconds);

    int justForWaiting;
    scanf("%d", &justForWaiting);

    return 0;
}